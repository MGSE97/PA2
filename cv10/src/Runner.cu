#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <time.h>
#include <math.h>

#define RAND false

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

const unsigned int N = 1 << 20;
const unsigned int MEMSIZE = N * sizeof(unsigned int);
const unsigned int NO_LOOPS = 1000;
const unsigned int THREAD_PER_BLOCK = 256;
const unsigned int GRID_SIZE = (N + THREAD_PER_BLOCK - 1)/THREAD_PER_BLOCK;

void fillData(unsigned int *data, const unsigned int length)
{
#if RAND
	srand(time(0));
	for (unsigned int i = 0; i < length; i++)
		data[i] = rand();
#else
	for (unsigned int i = 0; i < length; i++)
		data[i]= 1;
#endif
}

void printData(const unsigned int *data, const unsigned int length)
{
	if (data ==0) return;
	for (unsigned int i=0; i<length; i++)
	{
		printf("%u ", data[i]);
	}
}


__global__ void kernel(const unsigned int *a, const unsigned int *b, const unsigned int length, unsigned int *c)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid < length)
		c[tid] = a[tid] + b[tid];
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 1. - single stream, async calling </summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test1()
{
	unsigned int *a, *b, *c;
	unsigned int *da, *db, *dc;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE,hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipMalloc( (void**)&da, MEMSIZE );
	hipMalloc( (void**)&db, MEMSIZE );
	hipMalloc( (void**)&dc, MEMSIZE );

	// create stream
	hipStream_t stream;
	hipStreamCreate(&stream);

	unsigned int dataOffset = 0;

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);

	for(int i=0; i < NO_LOOPS; i++)
	{
		// Do copy kernel copy
		hipMemcpyAsync(da, a, MEMSIZE, hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(db, b, MEMSIZE, hipMemcpyHostToDevice, stream);
		kernel<<<GRID_SIZE, THREAD_PER_BLOCK, 0, stream>>>(da, db, N, dc);
		hipMemcpyAsync(c, dc, MEMSIZE, hipMemcpyDeviceToHost, stream);
		dataOffset += N;
	}

	// Wait for it and destroy
	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("\nTest time: %f ms\n", elapsedTime);

	printData(c, 100);
	
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 2. - two streams - depth first approach </summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test2()
{
	unsigned int* a, * b, * c;
	unsigned int* da0, * db0, * dc0;
	unsigned int* da1, * db1, * dc1;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipMalloc((void**)&da0, MEMSIZE);
	hipMalloc((void**)&db0, MEMSIZE);
	hipMalloc((void**)&dc0, MEMSIZE);

	hipMalloc((void**)&da1, MEMSIZE);
	hipMalloc((void**)&db1, MEMSIZE);
	hipMalloc((void**)&dc1, MEMSIZE);

	// create stream
	hipStream_t stream0;
	hipStreamCreate(&stream0);
	hipStream_t stream1;
	hipStreamCreate(&stream1);

	unsigned int dataOffset = 0;

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);

	for (int i = 0; i < NO_LOOPS; i += 2)
	{
		//stream 0
		hipMemcpyAsync(da0, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(db0, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream0);
		kernel<<<GRID_SIZE, THREAD_PER_BLOCK, 0, stream0>>>(da0, db0, N, dc0);
		hipMemcpyAsync(&c[dataOffset], dc0, MEMSIZE, hipMemcpyDeviceToHost, stream0);
		dataOffset += N;

		//stream 1
		hipMemcpyAsync(da1, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(db1, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream1);
		kernel<<<GRID_SIZE, THREAD_PER_BLOCK, 0, stream1>>>(da1, db1, N, dc1);
		hipMemcpyAsync(&c[dataOffset], dc1, MEMSIZE, hipMemcpyDeviceToHost, stream1);
		dataOffset += N;
	}

	// Wait for it and destroy
	hipStreamSynchronize(stream0);
	hipStreamDestroy(stream0);

	hipStreamSynchronize(stream1);
	hipStreamDestroy(stream1);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("\nTest time: %f ms\n", elapsedTime);

	printData(c, 100);

	hipFree(da0);
	hipFree(db0);
	hipFree(dc0);

	hipFree(da1);
	hipFree(db1);
	hipFree(dc1);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 3. - two streams - breadth first approach</summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test3()
{
	unsigned int* a, * b, * c;
	unsigned int* da0, * db0, * dc0;
	unsigned int* da1, * db1, * dc1;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipMalloc((void**)&da0, MEMSIZE);
	hipMalloc((void**)&db0, MEMSIZE);
	hipMalloc((void**)&dc0, MEMSIZE);

	hipMalloc((void**)&da1, MEMSIZE);
	hipMalloc((void**)&db1, MEMSIZE);
	hipMalloc((void**)&dc1, MEMSIZE);

	// create stream
	hipStream_t stream0;
	hipStreamCreate(&stream0);
	hipStream_t stream1;
	hipStreamCreate(&stream1);

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);
	fillData(b, NO_LOOPS * N);

	unsigned int dataOffset0 = 0;
	unsigned int dataOffset1 = N;
	for (int i = 0; i < NO_LOOPS; i += 2)
	{
		hipMemcpyAsync(da0, &a[dataOffset0], MEMSIZE, hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(da1, &a[dataOffset1], MEMSIZE, hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(db0, &b[dataOffset0], MEMSIZE, hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(db1, &b[dataOffset1], MEMSIZE, hipMemcpyHostToDevice, stream1);
		kernel<<<GRID_SIZE, THREAD_PER_BLOCK, 0, stream0>>>(da0, db0, N, dc0);
		kernel<<<GRID_SIZE, THREAD_PER_BLOCK, 0, stream1>>>(da1, db1, N, dc1);
		hipMemcpyAsync(&c[dataOffset0], dc0, MEMSIZE, hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(&c[dataOffset1], dc1, MEMSIZE, hipMemcpyDeviceToHost, stream1);
		dataOffset0 += 2 * N;
		dataOffset1 += 2 * N;
	}

	// Wait for it and destroy
	hipStreamSynchronize(stream0);
	hipStreamDestroy(stream0);

	hipStreamSynchronize(stream1);
	hipStreamDestroy(stream1);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("\nTest time: %f ms\n", elapsedTime);

	printData(c, 100);

	hipFree(da0);
	hipFree(db0);
	hipFree(dc0);

	hipFree(da1);
	hipFree(db1);
	hipFree(dc1);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);

	test1();
	test2();
	test3();

	return 0;
}
