#include "hip/hip_runtime.h"
// includes, cuda
#include <hip/hip_runtime.h>

#include <cudaDefs.h>
#include <imageManager.h>
#include <stdlib.h>

#include "imageKernels.cuh"

#define BLOCK_DIM 32
#define DATA_TYPE unsigned char

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

typedef struct image{
	DATA_TYPE* dData;
	unsigned int Width;
	unsigned int Height;
	unsigned int BPP;		//Bits Per Pixel = 8, 16, 2ColorToFloat_Channels, or 32 bit
	unsigned int Pitch;
} image_t;

KernelSetting ks;
KernelSetting ks2;

int* dResultsDataR = 0;
int* dResultsDataG = 0;
int* dResultsDataB = 0;
int* dResultsDataA = 0;

int* dResultsMaxR = 0;
int* dResultsMaxG = 0;
int* dResultsMaxB = 0;
int* dResultsMaxA = 0;

DATA_TYPE* dOutputDataR = 0;
DATA_TYPE* dOutputDataG = 0;
DATA_TYPE* dOutputDataB = 0;
DATA_TYPE* dOutputDataA = 0;

int3* dOutputMax = 0;

image loadSourceImage(const char* imageFileName)
{
	FreeImage_Initialise();
	FIBITMAP* tmp = ImageManager::GenericLoader(imageFileName, 0);
	if(FreeImage_GetBPP(tmp) != 32)
		tmp = FreeImage_ConvertTo32Bits(tmp);	// Large image fix

	image image;
	image.dData = 0;
	image.Width = FreeImage_GetWidth(tmp);
	image.Height = FreeImage_GetHeight(tmp);
	image.BPP = FreeImage_GetBPP(tmp);
	image.Pitch = FreeImage_GetPitch(tmp);		// FREEIMAGE align row data ... You have to use pitch instead of width
	//image.Pitch = image.Width * image.BPP / 8;

	checkCudaErrors(hipMallocManaged((void**)&image.dData, image.Pitch * image.Height));
	checkCudaErrors(hipMemcpy(image.dData, FreeImage_GetBits(tmp), image.Pitch * image.Height, hipMemcpyHostToDevice));

	//checkHostMatrix<DATA_TYPE>(FreeImage_GetBits(tmp), image.Pitch, image.Height, image.Width, "%hhu ", "Result of Linear Pitch Text");
	//checkDeviceMatrix<DATA_TYPE>(image.dData, image.Pitch, image.Height, image.Width, "%hhu ", "Result of Linear Pitch Text");

	FreeImage_Unload(tmp);
	//FreeImage_DeInitialise();

	return image;
}

void releaseMemory(image src)
{
	if (src.dData != 0)
		hipFree(src.dData);

	if (dResultsDataR)
		hipFree(dResultsDataR);
	if (dResultsDataG)
		hipFree(dResultsDataG);
	if (dResultsDataB)
		hipFree(dResultsDataB);
	if (dResultsDataA)
		hipFree(dResultsDataA);

	if (dOutputDataR)
		hipFree(dOutputDataR);
	if (dOutputDataG)
		hipFree(dOutputDataG);
	if (dOutputDataB)
		hipFree(dOutputDataB);
	if (dOutputDataA)
		hipFree(dOutputDataA);

	if (dOutputMax)
		hipFree(dOutputMax);

	FreeImage_DeInitialise();
}

// Set 0 in array
__global__ void zeroKernel(
	int* src, const unsigned int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < len)
		src[i] = 0;
}


// Compute max in 1D
__global__ void maxKernel(
	int* src, const unsigned int len, int* max)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < len)
		atomicMax(max, src[i]);
}

// Compute max and location in 2D
__global__ void max2DKernel(
	DATA_TYPE* src, const unsigned int channels,
	const unsigned int width, const unsigned int height, const unsigned int pitch,
	int3* result)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col <= width && row <= height)
	{
		int pixel = pitch / width;

		// Compute pixel color sum
		int c = 0;
		for (int i = 0; i < channels; i++)
			c += src[col * pixel + i + row * pitch];
		
		// Save max value
		atomicMax(&(result->z), c);
		
		// Save enought due to atomic operation in other threads and comparing to custom value
		if (result->z == c)
		{
			result->x = col;
			result->y = row;
		}
	}
}

// Sum histogram data
__global__ void histogramKernel(
	DATA_TYPE* src, const unsigned int channel,
	const unsigned int width, const unsigned int height, const unsigned int pitch,
	int* result)
{	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col <= width && row <= height)
	{
		int pixel = pitch / width;
		
		// Sum by color
		int c = src[col * pixel + channel + row * pitch];
		atomicAdd(&result[c], 1);
	}
}

// Convert histogram data to 2D chart
__global__ void histogram2DKernel(
	int* src, const unsigned int width, const unsigned int height, const int* limit, DATA_TYPE* result)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < width)
	{
		// Normalize src value to <0, 1>
		int v = src[x];
		double val = v / (double)*limit;
		// Convert to height part from bottom
		v = height - height * val;
		// Draw line
		for (int y = height - 1; y >= 0; y--)
			result[x  + y * width] = y >= v ? 255 : 0;
	}
}

void checkError(char* prefix)
{
	hipDeviceSynchronize();
	auto ex = hipGetLastError();
	if (ex != NULL)
		printf("Error at %s: %s\n", prefix, hipGetErrorString(ex));
}

// Save to file
void saveChannel(std::string name, const int size, const int limit, DATA_TYPE* data)
{
	BYTE* result = (BYTE*)malloc(size * limit);
	checkCudaErrors(hipMemcpy(result, data, size * limit, hipMemcpyDeviceToHost));

	//checkHostMatrix(result, size, limit, size, "%d ");
	FIBITMAP* img = FreeImage_ConvertFromRawBits(result, limit, limit, size, 8, 0xFF, 0xFF, 0xFF);
	FreeImage_FlipVertical(img);
	
	//ToDo: Edit to custom location
	ImageManager::GenericWriter(img, ("D:\\Documents\\Projekty\\�kola\\PA2\\cv9\\assets\\"+name+".png").c_str(), 0);

	FreeImage_Unload(img);
	SAFE_DELETE(result);
}

int main(int argc, char* argv[])
{
	initializeCUDA(deviceProp);

	//ToDo: Edit to custom location
	image src = loadSourceImage("D:\\Documents\\Projekty\\�kola\\PA2\\cv9\\assets\\lena.png");
	//image src = loadSourceImage("D:\\Documents\\Projekty\\�kola\\PA2\\cv9\\assets\\RGB.png");
	//image src = loadSourceImage("D:\\Documents\\Projekty\\�kola\\PA2\\cv9\\assets\\dot.png");


	printf("Loaded\n");

	ks.dimBlock = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	ks.blockSize = BLOCK_DIM * BLOCK_DIM;
	ks.dimGrid = dim3((src.Width + BLOCK_DIM - 1) / BLOCK_DIM, (src.Height + BLOCK_DIM - 1) / BLOCK_DIM, 1);
	
	ks2.dimBlock = dim3(255, 1, 1);
	ks2.blockSize = 255;
	ks2.dimGrid = dim3(1, 1, 1);

	const int ch_size = 255 * sizeof(char);
	const int ch_limit = 100;
	const int cmp_size = 255 * sizeof(int);
	hipMallocManaged((void**)&dResultsDataR, cmp_size);
	hipMallocManaged((void**)&dResultsDataG, cmp_size);
	hipMallocManaged((void**)&dResultsDataB, cmp_size);
	hipMallocManaged((void**)&dResultsDataA, cmp_size);

	hipMallocManaged((void**)&dResultsMaxR, sizeof(int));
	hipMallocManaged((void**)&dResultsMaxG, sizeof(int));
	hipMallocManaged((void**)&dResultsMaxB, sizeof(int));
	hipMallocManaged((void**)&dResultsMaxA, sizeof(int));

	hipMallocManaged((void**)&dOutputDataR, ch_size * ch_limit);
	hipMallocManaged((void**)&dOutputDataG, ch_size * ch_limit);
	hipMallocManaged((void**)&dOutputDataB, ch_size * ch_limit);
	hipMallocManaged((void**)&dOutputDataA, ch_size * ch_limit);

	hipMallocManaged((void**)&dOutputMax, sizeof(int3));
	
	checkError("Malloc");
	hipEvent_t start, stop;
	float time;
	createTimer(&start, &stop, &time);

	startTimer(start);
	zeroKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataR, 255); checkError("Z-R");
	zeroKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataG, 255); checkError("Z-G");
	zeroKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataB, 255); checkError("Z-B");
	zeroKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataA, 255); checkError("Z-A");

	histogramKernel<<<ks.dimGrid, ks.dimBlock>>>(src.dData, 2, src.Width, src.Height, src.Pitch, dResultsDataR); checkError("R");
	histogramKernel<<<ks.dimGrid, ks.dimBlock>>>(src.dData, 1, src.Width, src.Height, src.Pitch, dResultsDataG); checkError("G");
	histogramKernel<<<ks.dimGrid, ks.dimBlock>>>(src.dData, 0, src.Width, src.Height, src.Pitch, dResultsDataB); checkError("B");
	histogramKernel<<<ks.dimGrid, ks.dimBlock>>>(src.dData, 3, src.Width, src.Height, src.Pitch, dResultsDataA); checkError("A");


	maxKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataR, 255, dResultsMaxR); checkError("M-R");
	maxKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataG, 255, dResultsMaxG); checkError("M-G");
	maxKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataB, 255, dResultsMaxB); checkError("M-B");
	maxKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataA, 255, dResultsMaxA); checkError("M-A");

	histogram2DKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataR, 255, ch_limit, dResultsMaxR, dOutputDataR); checkError("2D-R");
	histogram2DKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataG, 255, ch_limit, dResultsMaxG, dOutputDataG); checkError("2D-G");
	histogram2DKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataB, 255, ch_limit, dResultsMaxB, dOutputDataB); checkError("2D-B");
	histogram2DKernel<<<ks2.dimGrid, ks2.blockSize>>>(dResultsDataA, 255, ch_limit, dResultsMaxA, dOutputDataA); checkError("2D-A");

	max2DKernel<<<ks.dimGrid, ks.dimBlock>>>(src.dData, 4, src.Width, src.Height, src.Pitch, dOutputMax); checkError("Max-2D");

	stopTimer(start, stop, time);
	printf("Time: %f ms\n", time);

	saveChannel("R", ch_size, ch_limit, dOutputDataR);
	saveChannel("G", ch_size, ch_limit, dOutputDataG);
	saveChannel("B", ch_size, ch_limit, dOutputDataB);
	saveChannel("A", ch_size, ch_limit, dOutputDataA);
	
	// Historograms are scaled by maximal value on channel
	int4 maxs;
	hipMemcpy(&maxs.x, dResultsMaxR, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&maxs.y, dResultsMaxG, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&maxs.z, dResultsMaxB, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&maxs.w, dResultsMaxA, sizeof(int), hipMemcpyDeviceToHost);
	printf("Historogram scaling:\nMAX: R: %d, G: %d, B: %d, A: %d\n", maxs.x, maxs.y, maxs.z, maxs.w);

	int3 max;
	hipMemcpy(&max, dOutputMax, sizeof(int3), hipMemcpyDeviceToHost);
	printf("Most exposed pixel:\nX:\t%d\nY:\t%d\nSum:\t%d\n", max.x, src.Height - max.y, max.z);

	releaseMemory(src);
}
